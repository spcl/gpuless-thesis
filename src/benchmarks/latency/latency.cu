
#include <hip/hip_runtime.h>
#include <chrono>
#include <iostream>

int main() {
    auto s = std::chrono::high_resolution_clock::now();
    float x = 1.0;
    float *d_ptr;
    hipMalloc(&d_ptr, sizeof(float));
    hipMemcpy(d_ptr, &x, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(&x, d_ptr, sizeof(float), hipMemcpyDeviceToHost);
    auto e = std::chrono::high_resolution_clock::now();
    auto d = std::chrono::duration_cast<std::chrono::microseconds>(e-s).count() / 1000000.0;
    printf("%.10f\n", d);
    return 0;
}
