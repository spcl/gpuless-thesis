#include <cassert>
#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>

#define N 4096

__global__ void vadd(const float *a, const float *b, float *c, int n) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) {
        c[i] = a[i] + b[i];
    }
}

int main() {
    float *h_a = (float *)malloc(N * sizeof(float));
    float *h_b = (float *)malloc(N * sizeof(float));
    float *h_c = (float *)malloc(N * sizeof(float));

    for (int i = 0; i < N; i++) {
        h_a[i] = i;
        h_b[i] = i;
    }

    float *d_a, *d_b, *d_c;
    hipMalloc(&d_a, 2 * N * sizeof(float));
    hipMalloc(&d_b, 2 * N * sizeof(float));
    hipMalloc(&d_c, N * sizeof(float));

    hipMemcpy(d_a + N, h_a, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b + N, h_b, N * sizeof(float), hipMemcpyHostToDevice);

    //    const int n = N;
    //    void *args[] = {&d_a, &d_b, &d_c, (void *)&n};
    //    cudaLaunchKernel((const void *)vadd, dim3(N), dim3(1), args, 0, 0);
    vadd<<<N, 1>>>(d_a + N, d_b + N, d_c, N);
    vadd<<<N, 1>>>(d_c, d_a + N, d_c, N);
    vadd<<<N, 1>>>(d_c, d_a + N, d_c, N);
    vadd<<<N, 1>>>(d_c, d_a + N, d_c, N);

    hipMemcpy(h_c, d_c, N * sizeof(float), hipMemcpyDeviceToHost);

    printf("%f\n", h_c[0]);
    printf("%f\n", h_c[1]);
    printf("%f\n", h_c[2]);
    printf("%f\n", h_c[3]);
    printf("%f\n", h_c[4]);

    assert(abs(h_c[0] - 0.0) < 1e-6);
    assert(abs(h_c[1] - 5.0) < 1e-6);
    assert(abs(h_c[2] - 10.0) < 1e-6);
    assert(abs(h_c[3] - 15.0) < 1e-6);
    assert(abs(h_c[4] - 20.0) < 1e-6);

    return 0;
}
