
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <cassert>
#include <cudnn.h>
#include <cstdint>

struct T {
    uint64_t a;
    int* b;
};

__global__ void class_kernel(T test, uint64_t* c) {
    *c = test.a + *test.b;
}

int main() {
    int *d_b;
    uint64_t *d_c;

    int b = 2;
    hipMalloc(&d_b, sizeof(int));
    hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice);

    T test_c{1UL, d_b};

    hipMalloc(&d_c, sizeof(uint64_t));
    class_kernel<<<1, 1>>>(test_c, d_c);
    uint64_t result;
    hipMemcpy(&result, d_c, sizeof(uint64_t), hipMemcpyDeviceToHost);

    hipFree(d_b);
    hipFree(d_c);
    
    printf("%lu\n", result);

    assert(result == 3);

    return 0;
}
