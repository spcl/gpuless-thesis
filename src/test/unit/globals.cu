
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <cassert>

#define N 4096

__device__ float dev_const = 420.0;

__global__ void vadd(const float *a, const float *b, float *c, int n) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) {
        c[i] = a[i] + b[i] + dev_const;
    }
}

int main() {
    float *h_a = (float *)malloc(N * sizeof(float));
    float *h_b = (float *)malloc(N * sizeof(float));
    float *h_c = (float *)malloc(N * sizeof(float));

    for (int i = 0; i < N; i++) {
        h_a[i] = i;
        h_b[i] = i;
    }

    float *d_a, *d_b, *d_c;
    hipMalloc(&d_a, N * sizeof(float));
    hipMalloc(&d_b, N * sizeof(float));
    hipMalloc(&d_c, N * sizeof(float));

    hipMemcpy(d_a, h_a, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, N * sizeof(float), hipMemcpyHostToDevice);
    vadd<<<N, 1>>>(d_a, d_b, d_c, N);
    hipMemcpy(h_c, d_c, N * sizeof(float), hipMemcpyDeviceToHost);

    assert(abs(h_c[0] - (420.0 + 0.0)) < 1e-6);
    assert(abs(h_c[1] - (420.0 + 2.0)) < 1e-6);
    assert(abs(h_c[2] - (420.0 + 4.0)) < 1e-6);
    assert(abs(h_c[3] - (420.0 + 6.0)) < 1e-6);
    assert(abs(h_c[4] - (420.0 + 8.0)) < 1e-6);

    free(h_a);
    free(h_b);
    free(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
